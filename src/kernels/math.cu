
#include <hip/hip_runtime.h>
// Mathematical operations kernels
extern "C" {

// Exponential function
__global__ void exp_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = expf(input[idx]);
    }
}

// Natural logarithm
__global__ void log_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = logf(input[idx]);
    }
}

// Square root
__global__ void sqrt_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = sqrtf(input[idx]);
    }
}

// Power function (input^power)
__global__ void pow_kernel(const float* input, float* output, float power, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = powf(input[idx], power);
    }
}

// Sine function
__global__ void sin_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = sinf(input[idx]);
    }
}

// Cosine function
__global__ void cos_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = cosf(input[idx]);
    }
}

// sinh function
__global__ void sinh_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = sinhf(input[idx]);
    }
}

// cosh function
__global__ void cosh_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = coshf(input[idx]);
    }
}

// ReLU activation function
__global__ void relu_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

// ReLU backward function
__global__ void mask_for_grad_relu_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = (float)(input[idx]> 0.0f);
    }
}

// Sigmoid activation function
__global__ void sigmoid_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = 1.0f / (1.0f + expf(-input[idx]));
    }
}

// Tanh activation function
__global__ void tanh_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = tanhf(input[idx]);
    }
}


__global__ void clamp_max_kernel(const float* input, float* output, float max, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fmaxf(input[idx],max);
    }
}

__global__ void clamp_min_kernel(const float* input, float* output, float min, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fminf(input[idx], min);
    }
}


} // extern "C"